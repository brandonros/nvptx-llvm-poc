#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../output/ptx_code.h"

int main() {
    hipDevice_t device;
    hipCtx_t context;
    hipModule_t module;
    hipFunction_t function;

    printf("Initializing CUDA\n");
    hipInit(0);
    hipDeviceGet(&device, 0);
    hipCtxCreate(&context, 0, device);

    printf("Loading PTX\n");
    hipModuleLoadData(&module, ptx_code);
    hipModuleGetFunction(&function, module, "add");

    // Allocate GPU memory
    printf("Allocating GPU memory\n");
    hipDeviceptr_t d_result;
    hipMalloc(&d_result, sizeof(int));
    
    int a = 2, b = 2;
    
    // Pass the device pointer directly, not pointer to it
    void* args[] = { &d_result, &a, &b };

    printf("Launching kernel\n");
    hipModuleLaunchKernel(function, 1, 1, 1, 1, 1, 1, 0, NULL, args, NULL);
    hipCtxSynchronize();

    int result;
    hipMemcpyDtoH(&result, d_result, sizeof(int));
    printf("Result: %d\n", result);

    hipFree(d_result);
    return 0;
}